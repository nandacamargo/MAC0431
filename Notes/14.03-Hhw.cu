#include "hip/hip_runtime.h"
#include "util/cuPrintf.cu"
#include <stdio.h>
#include <stdlib.h>

/**
 * For all functions, we can have 3 different access levels:
 * __host__   : function to be executed by the host, default.
 * __global__ : function to be executed by the device (GPU),
 *              but can be called by the host.
 * __device__ : function to be executed only by the device,
 *              not accessible from the host.
 */

__global__ void device_greetings(void)
{
    cuPrintf("Hello, world from the device! (%d, %d) (%d, %d)",
            blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

int main(int argc, char **argv) {
  int Gx = 1, Gy = 1, Bx = 1, By = 1;

  printf("Hello, world from the host");

  // Initialize printf inside device
  cudaPrintfInit();

  // Grade dimensions
  dim3 Grade(Gx, Gy);
  dim3 Blocks(Bx, By);

  // Execute function for Grade and Block
  device_greetings<<<Grade, Blocks>>>();

  // Display in the host output from device
  cudaPrintfDisplay();

  // End output from device
  cudaPrintfEnd();

  return 0;
}
