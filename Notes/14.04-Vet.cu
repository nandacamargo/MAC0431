#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

long int TAM_VET, VEZES;
/* Um vetor para o host, outro para a placa */
double *vet, *vet_d;

__global__ void calcula(double *vet_d, long int TAM_VET) {
    long int idx = blockIdx.x * blockDim.x + threadIdx.x;

    /* Para não passar dos limites */
    if (idx >= TAM_VET) return;
    vet_d[idx] += cos(idx * sqrt(idx * 1.0) * 4.0);
}

int main(int argc, char **argv) {
  int B = 1000, T = 1024; /* Blocos e threads */

  if ((vet = malloc(TAM_VET * sizeof(double))) != NULL) {
    fprintf(stderr, "Could not alloc block of size %d in the host",
            TAM_VET * sizeof(double));
  }

  if (hipMalloc(&vet_d, TAM_VET * sizeof(double)) != hipSuccess) {
    fprintf(stderr, "Could not alloc block of size %d in the device",
            TAM_VET * sizeof(double));
  }

  if (hipMemcpy(vet_d, vet, TAM_VET*sizeof(double),
      hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Could not copy block of size %d in the device",
            TAM_VET * sizeof(double));
  }

  hipFree(vet_d);
  free(vet);
}
